#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include "cuda_spmm.h"
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#define FTYPE float

using namespace sputnik;

using namespace std;
#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (yHostPtr)           free(yHostPtr);          \
    if (zHostPtr)           free(zHostPtr);          \
    if (xIndHostPtr)        free(xIndHostPtr);       \
    if (xValHostPtr)        free(xValHostPtr);       \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
    if (y)                  hipFree(y);             \
    if (z)                  hipFree(z);             \
    if (xInd)               hipFree(xInd);          \
    if (xVal)               hipFree(xVal);          \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
    if (cooRowIndex)        hipFree(cooRowIndex);   \
    if (cooColIndex)        hipFree(cooColIndex);   \
    if (cooVal)             hipFree(cooVal);        \
    if (handle)             hipsparseDestroy(handle); \
    fflush (stdout);                                 \
} while (0)

struct v_struct {
        int row, col;
        FTYPE val;
};

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

int compare1(const void *a, const void *b)
{
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row > 0) return 1;
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row < 0) return -1;
        return ((struct v_struct *)a)->col - ((struct v_struct *)b)->col;
}


int main(int argc, char **argv)
{

    if(argc < 4){
        printf("Wrong arg list. Try with: ./exec matrix rhs nBlocks.\n"); 
        printf("E.g., ./spmm_blocked tmp.mtx 32 1 \n"); 
        exit(0);
    }

	FILE *fp;
	FILE *fpo = fopen("SpMM_GPU_SP_spmm.out", "a");
	srand(time(NULL));

    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descra=0;

    int n, nrows, ncols, nnz, nflag, nnz_vector, i, j;

	struct v_struct *temp_v;

	char buf[300];
	int sflag;
	int dummy, pre_count=0, tmp_ne;
    int rhs = atoi(argv[2]);   
    int nBlock = atoi(argv[3]);
	fp = fopen(argv[1], "r");

	fgets(buf, 300, fp);

    if(strstr(buf, "symmetric") != NULL || strstr(buf, "Hermitian") != NULL) sflag = 1; // symmetric
    else sflag = 0;
    if(strstr(buf, "pattern") != NULL) nflag = 0; // non-value
    else if(strstr(buf, "complex") != NULL) nflag = -1;
    else nflag = 1;

    while(1) {
            pre_count++;
            fgets(buf, 300, fp);
            if(strstr(buf, "%") == NULL) break;
    }
    fclose(fp);

    fp = fopen(argv[1], "r");
    for(i=0;i<pre_count;i++)
            fgets(buf, 300, fp);

    fscanf(fp, "%d %d %d", &nrows, &ncols, &nnz);
    nnz *= (sflag+1);

    temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(nnz+1));

    /*------------------------
      Read input matrix
    ------------------------*/
    for(i=0;i<nnz;i++) {
        fscanf(fp, "%d %d", &temp_v[i].row, &temp_v[i].col);
        temp_v[i].row--; temp_v[i].col--;

        if(temp_v[i].row < 0 || temp_v[i].row >= nrows || temp_v[i].col < 0 || temp_v[i].col >= ncols) {
                fprintf(stdout, "A vertex id is out of range %d %d\n", temp_v[i].row, temp_v[i].col);
                exit(0);
        }
        if(nflag == 0) temp_v[i].val = (FTYPE)(rand()%1048576)/1048576;
        else if(nflag == 1) {
                FTYPE ftemp;
                fscanf(fp, " %f ", &ftemp);
                temp_v[i].val = ftemp;
        } else { // complex
                FTYPE ftemp1, ftemp2;
                fscanf(fp, " %f %f ", &ftemp1, &ftemp2);
                temp_v[i].val = ftemp1;
        }
        if(sflag == 1) {
                i++;
                temp_v[i].row = temp_v[i-1].col;
                temp_v[i].col = temp_v[i-1].row;
                temp_v[i].val = temp_v[i-1].val;
        }
    }
    qsort(temp_v, nnz, sizeof(struct v_struct), compare1);

    /*------------------------
      Estimate block nnz
    ------------------------*/  
    int p = nBlock;
    int rootp = sqrt(nBlock);

    int nRowsBlock = (nrows + rootp - 1) / rootp;
    // int nColsBlock = (ncols + rootp - 1) / rootp; 
    int b_rhs = (rhs + rootp -1)/rootp;
    int *nnzBlock = (int*)malloc(rootp * sizeof(int));    
    memset(nnzBlock, 0, rootp * sizeof(int));
    int bi, bj, bid;

    for(i = 0; i < nnz; i++){
        bi = temp_v[i].row/nRowsBlock;
        // bj = temp_v[i].col/nColsBlock;
        bid = bi;// * rootp + bj;
        nnzBlock[bid]++;  
    }
    int tot_nnz = 0;
    
    /*------------------------
      Created blocked COO/CSR
    ------------------------*/  
    int **b_rowPtr = (int **)malloc(rootp * sizeof (int *) );
    int **b_rowInd = (int **)malloc(rootp * sizeof (int *) ); //sptnk uses random
    int **b_colInd = (int **)malloc(rootp * sizeof (int *) );
    FTYPE **b_val = (FTYPE **)malloc(rootp * sizeof (FTYPE *) );

    for (int b = 0; b < rootp; b++) {
        b_rowPtr[b] = (int *) malloc ((nRowsBlock+1) * sizeof (int)) ;
        memset(&b_rowPtr[b][0], 0, (nRowsBlock+1) * sizeof(int));
        b_rowInd[b] = (int *) malloc (nRowsBlock * sizeof (int)) ;
        b_colInd[b] = (int *) malloc ((nnzBlock[b]) * sizeof (int)) ;
        b_val[b] = (FTYPE  *) malloc ((nnzBlock[b]) * sizeof (FTYPE   )) ;
    }
    memset( nnzBlock, 0, (rootp) * sizeof(int) );

    for(i = 0; i < nnz; i++){
        int br = temp_v[i].row/nRowsBlock;
        // int bc = temp_v[i].col/nColsBlock;
        int bId = br;// * rootp + bc;
        
        int local_rowInd = temp_v[i].row % nRowsBlock;
        b_rowPtr[bId][1+local_rowInd] = nnzBlock[bId]+1;
        b_rowInd[bId][nnzBlock[bId]] = temp_v[i].row % nRowsBlock;
        b_colInd[bId][nnzBlock[bId]] = temp_v[i].col;// % nColsBlock;
        b_val[bId][nnzBlock[bId]] = temp_v[i].val;
        nnzBlock[bId]++;
    }

    for (int b = 0; b < rootp; ++b){
        for(int r = 0; r < nRowsBlock; r++) {
            b_rowInd[b][i] = r;//
            if(b_rowPtr[b][r] == 0)
                b_rowPtr[b][r] = b_rowPtr[b][r-1];
        }   
    }

    /*------------------------
      Created blocked CSR
    ------------------------*/  

    //correctness check of blocks

    float tot_time = 0;

    for (int br = 0; br < rootp; ++br) //loop over row blocks of C 
    {
        for (int bc = 0; bc < rootp; ++bc) //loop over col blocks C     
        {         
            int b = br;// * rootp + bc;
            if(!nnzBlock[b]) continue;  

            nnz = nnzBlock[b];
            nrows = nRowsBlock;
            
            /* copy CSR to GPU*/
            int *d_rowPtr, *d_rowInd, *d_colInd; 
            FTYPE *d_vals;
            hipMalloc((void **) &d_rowPtr, sizeof(int)*(nrows+1));
            hipMalloc((void **) &d_rowInd, sizeof(int)*(nrows));
            hipMalloc((void **) &d_colInd, sizeof(int)*nnz);
            hipMalloc((void **) &d_vals, sizeof(FTYPE)*nnz);
            hipMemcpy(d_rowPtr, &(b_rowPtr[b][0]), sizeof(int)*(nrows+1), hipMemcpyHostToDevice);
            hipMemcpy(d_rowInd, &(b_rowInd[b][0]), sizeof(int)*(nrows), hipMemcpyHostToDevice);
            hipMemcpy(d_colInd, &(b_colInd[b][0]), sizeof(int) * nnz, hipMemcpyHostToDevice);
            hipMemcpy(d_vals, &(b_val[b][0]), sizeof(FTYPE) * nnz, hipMemcpyHostToDevice);
            
            /* copy  dense input  and output matrices to GPU*/
        	hipError_t err = hipSuccess;
        	FTYPE *cy_in, *cy_out; 
        	FTYPE *y_in = (FTYPE *) malloc( sizeof(FTYPE) * ncols * b_rhs);
        	FTYPE *y_out = (FTYPE *) malloc( sizeof(FTYPE)* nrows * b_rhs);
        	
            for(int i=0; i < ncols * b_rhs; i++)
        		y_in[i] = ((FTYPE)1);//(rand()%1048576))/1048576;

        	hipMalloc((void **) &cy_in, sizeof(FTYPE) * ncols * b_rhs);
        	hipMalloc((void **) &cy_out, sizeof(FTYPE)*(nrows)*b_rhs);
        	hipMemcpy(cy_in, y_in, sizeof(FTYPE)*ncols*b_rhs, hipMemcpyHostToDevice);
            hipMemset((void *)cy_out, 0, b_rhs*(nrows)*sizeof(FTYPE));    

        	float tot_ms;
            hipEvent_t event1, event2;
            hipEventCreate(&event1);
            hipEventCreate(&event2);
            
            hipDeviceSynchronize();
            hipEventRecord(event1,0);
            
            #define ITER (10)
            for(int ik=0;ik<ITER;ik++) {
                // CudaSpmm(nrows, ncols, b_rhs, nnz,
                //          d_rowInd,
                //          d_vals,
                //          d_rowPtr,
                //          d_colInd,
                //          cy_in,
                //          cy_out, nullptr);
            }

            hipEventRecord(event2,0);
            hipEventSynchronize(event1);
            hipEventSynchronize(event2);
            hipEventElapsedTime(&tot_ms, event1, event2);
            hipDeviceSynchronize();

            if (status != HIPSPARSE_STATUS_SUCCESS) return EXIT_FAILURE;
    	    hipMemcpy(y_out, cy_out, sizeof(FTYPE) * nrows * b_rhs, hipMemcpyDeviceToHost);

         	hipFree(d_rowPtr); hipFree(d_rowInd); hipFree(d_colInd); hipFree(d_vals); 
            hipFree(cy_out); hipFree(cy_in); free(y_out); free(y_in);
            tot_time += tot_ms;
            fprintf(stdout, "Block: %d, nnz: %d, tot_ms: %f s\n", b, nnzBlock[b], tot_ms/ITER);
        }
    }

    for (int b = 0; b < rootp; ++b){
        free(b_rowPtr[b]); free(b_colInd[b]); free(b_val[b]);
    }
    fprintf(stdout, "1D Blocking, K=%d : nBlocks: %d, nnz: %d, tot_ms: %f ms, GFLOPS: %f \n", rhs, nBlock, nnz, tot_time, (double)ITER*(double)nnz*2*rhs/tot_time/1000000);
	fclose(fpo);     
}


