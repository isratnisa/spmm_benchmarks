#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#define FTYPE float

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    if (yHostPtr)           free(yHostPtr);          \
    if (zHostPtr)           free(zHostPtr);          \
    if (xIndHostPtr)        free(xIndHostPtr);       \
    if (xValHostPtr)        free(xValHostPtr);       \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
    if (y)                  hipFree(y);             \
    if (z)                  hipFree(z);             \
    if (xInd)               hipFree(xInd);          \
    if (xVal)               hipFree(xVal);          \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
    if (cooRowIndex)        hipFree(cooRowIndex);   \
    if (cooColIndex)        hipFree(cooColIndex);   \
    if (cooVal)             hipFree(cooVal);        \
    if (handle)             hipsparseDestroy(handle); \
    fflush (stdout);                                 \
} while (0)

struct v_struct {
        int row, col;
        FTYPE val;
};

int *csr_v, *csr_colIdx;
FTYPE *csr_vals;

double rtclock(void)
{
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday (&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d",stat);
  return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

int compare1(const void *a, const void *b)
{
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row > 0) return 1;
        if (((struct v_struct *)a)->row - ((struct v_struct *)b)->row < 0) return -1;
        return ((struct v_struct *)a)->col - ((struct v_struct *)b)->col;
}


int main(int argc, char **argv)
{

    if(argc < 4){
        printf("Wrong arg list. Try with: ./exec matrix rhs nBlocks.\n"); 
        printf("E.g., ./spmm_blocked tmp.mtx 32 1 \n"); 
        exit(0);
    }

	FILE *fp;
	FILE *fpo = fopen("SpMM_GPU_SP_spmm.out", "a");
	srand(time(NULL));

    hipError_t cudaStat1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descra=0;

    int n, nr, nc, nnz, nflag, nnz_vector, i, j;

	struct v_struct *temp_v;

	char buf[300];
	int sflag;
	int dummy, pre_count=0, tmp_ne;
    int sc = atoi(argv[2]);   
    int nBlock = atoi(argv[3]);
	fp = fopen(argv[1], "r");

	fgets(buf, 300, fp);

    if(strstr(buf, "symmetric") != NULL || strstr(buf, "Hermitian") != NULL) sflag = 1; // symmetric
    else sflag = 0;
    if(strstr(buf, "pattern") != NULL) nflag = 0; // non-value
    else if(strstr(buf, "complex") != NULL) nflag = -1;
    else nflag = 1;

    while(1) {
            pre_count++;
            fgets(buf, 300, fp);
            if(strstr(buf, "%") == NULL) break;
    }
    fclose(fp);

    fp = fopen(argv[1], "r");
    for(i=0;i<pre_count;i++)
            fgets(buf, 300, fp);

    fscanf(fp, "%d %d %d", &nr, &nc, &nnz);
    nnz *= (sflag+1);

    temp_v = (struct v_struct *)malloc(sizeof(struct v_struct)*(nnz+1));

    /*------------------------
      Read input matrix
    ------------------------*/
    for(i=0;i<nnz;i++) {
            fscanf(fp, "%d %d", &temp_v[i].row, &temp_v[i].col);
            temp_v[i].row--; temp_v[i].col--;

            if(temp_v[i].row < 0 || temp_v[i].row >= nr || temp_v[i].col < 0 || temp_v[i].col >= nc) {
                    fprintf(stdout, "A vertex id is out of range %d %d\n", temp_v[i].row, temp_v[i].col);
                    exit(0);
            }
            if(nflag == 0) temp_v[i].val = (FTYPE)(rand()%1048576)/1048576;
            else if(nflag == 1) {
                    FTYPE ftemp;
                    fscanf(fp, " %f ", &ftemp);
                    temp_v[i].val = ftemp;
            } else { // complex
                    FTYPE ftemp1, ftemp2;
                    fscanf(fp, " %f %f ", &ftemp1, &ftemp2);
                    temp_v[i].val = ftemp1;
            }
            if(sflag == 1) {
                    i++;
                    temp_v[i].row = temp_v[i-1].col;
                    temp_v[i].col = temp_v[i-1].row;
                    temp_v[i].val = temp_v[i-1].val;
            }
    }
    qsort(temp_v, nnz, sizeof(struct v_struct), compare1);

    /*------------------------
      Estimate block nnz
    ------------------------*/  
    int p = nBlock;
    int rootp = sqrt(nBlock);

    int nRowsBlock = (nr + rootp - 1) / rootp;
    int nColsBlock = (nc + rootp - 1) / rootp; 
    sc = (sc + rootp -1)/rootp;
    int *nnzBlock = (int*)malloc(nBlock * sizeof(int));    
    memset(nnzBlock, 0, nBlock * sizeof(int));
    int bi, bj, bid;

    for(i = 0; i < nnz; i++){
        bi = temp_v[i].row/nRowsBlock;
        bj = temp_v[i].col/nColsBlock;
        bid = bi * rootp + bj;
        nnzBlock[bid]++;  
    }
    int tot_nnz = 0;
    //correctness check of blocks
    // for (int b = 0; b < nBlock; ++b) {tot_nnz += nnzBlock[b];
    // printf("nnzBlock %d %d\n", nnzBlock[b], tot_nnz);}


    // loop through blocks of each benchmark
    for (int br = 0; br < rootp; ++br) {//loop over row blocks of C 
        for (int bc = 0; bc < rootp; ++bc) {//loop over col blocks C          
            for (int bk = 0; bk < rootp; ++bk) {// loop over blocks of A & B   
                int b = br * rootp + bk;
                int bb = bk * rootp + bc; //doesnt matter here..all sublcoks are same 
                // printf("Processing block %d %d %d - nnz:  %d\n", br, bc, bk, nnzBlock[b] );
                if(!nnzBlock[b]) continue;  

                nnz = nnzBlock[b];
                nr = nRowsBlock;
                nc = nColsBlock;

                int nrows_ = nr;
                int nvals_ = nnz;
                int *h_csrRowPtr_ = (int *)malloc(sizeof(int)*(nr+1));
                // csr_v = (int *)malloc(sizeof(int)*(nr+1));
                csr_colIdx = (int *)malloc(sizeof(int)*nnz);
                csr_vals = (FTYPE *)malloc(sizeof(FTYPE)*nnz);

                // Convert to CSR/CSC
                int temp, row, col, dest, cumsum=0;

                // Set all rowPtr to 0
                for( int i=0; i<=nrows_; i++ )
                  h_csrRowPtr_[i] = 0;
                // Go through all elements to see how many fall in each row
                for( int i=0; i<nvals_; i++ ) {
                  row = temp_v[i].row % nRowsBlock;
                  // if( row>=nrows_ ) return GrB_INDEX_OUT_OF_BOUNDS;
                  h_csrRowPtr_[ row ]++;
                }
                // Cumulative sum to obtain rowPtr
                for( int i=0; i<nrows_; i++ ) {
                  temp = h_csrRowPtr_[i];
                  h_csrRowPtr_[i] = cumsum;
                  cumsum += temp;
                }
                h_csrRowPtr_[nrows_] = nvals_;

                // Store colInd and val
                for( int i=0; i<nvals_; i++ ) {
                  row = temp_v[i].row % nRowsBlock;
                  dest= h_csrRowPtr_[row];
                  col = temp_v[i].col % nColsBlock;
                  // if( col>=ncols_ ) return GrB_int_OUT_OF_BOUNDS;
                  csr_colIdx[dest] = col;
                  csr_vals[dest]    = temp_v[i].val;
                  h_csrRowPtr_[row]++;
                }
                cumsum = 0;
                
                // Undo damage done to rowPtr
                for( int i=0; i<=nrows_; i++ ) {
                  temp = h_csrRowPtr_[i];
                  h_csrRowPtr_[i] = cumsum;
                  cumsum = temp;
                }
            	
                int *ccsr_v, *ccsr_e; FTYPE *ccsr_ev;

            	hipMalloc((void **) &ccsr_v, sizeof(int)*(nr+1));
            	hipMalloc((void **) &ccsr_e, sizeof(int)*nnz);
            	hipMalloc((void **) &ccsr_ev, sizeof(FTYPE)*nnz);
            	hipMemcpy(ccsr_v, h_csrRowPtr_, sizeof(int)*(nr+1), hipMemcpyHostToDevice);
            	hipMemcpy(ccsr_e, csr_colIdx, sizeof(int)*(nnz), hipMemcpyHostToDevice);
            	hipMemcpy(ccsr_ev, csr_vals, sizeof(FTYPE)*(nnz), hipMemcpyHostToDevice);
            	

                /* initialize cusparse library */
                status= hipsparseCreate(&handle);
                if (status != HIPSPARSE_STATUS_SUCCESS) {
                    return EXIT_FAILURE;
                }
                /* create and setup matrix descriptor */ 
                status= hipsparseCreateMatDescr(&descra); 
                if (status != HIPSPARSE_STATUS_SUCCESS) {
                    return EXIT_FAILURE;
                }       
                hipsparseSetMatType(descra,HIPSPARSE_MATRIX_TYPE_GENERAL);
                hipsparseSetMatIndexBase(descra,HIPSPARSE_INDEX_BASE_ZERO);  

            	hipError_t err = hipSuccess;
            	FTYPE *y_in, *cy_in, *y_out, *cy_out; 
            	y_in = (FTYPE *)malloc(sizeof(FTYPE)*nc*sc);
            	y_out = (FTYPE *)malloc(sizeof(FTYPE)*(nr)*sc);
            	
                for(int i=0;i<nc*sc;i++)
            		y_in[i] = ((FTYPE)1);//(rand()%1048576))/1048576;

            	err = hipMalloc((void **) &cy_in, sizeof(FTYPE)*nc*sc);
                    if(err != hipSuccess)  {fprintf(stdout, "\n"); exit(0); }
            	err = hipMalloc((void **) &cy_out, sizeof(FTYPE)*(nr)*sc);
                    if(err != hipSuccess)  {fprintf(stdout, "\n"); exit(0); }
            	hipMemcpy(cy_in, y_in, sizeof(FTYPE)*nc*sc, hipMemcpyHostToDevice);
                	hipMemset((void *)cy_out, 0, sc*(nr)*sizeof(FTYPE));    

            	float tot_ms;
                hipEvent_t event1, event2;
                hipEventCreate(&event1);
                hipEventCreate(&event2);

            	const FTYPE alpha=1.0f, beta=0.0f;

                /*new SpMM*/

                hipsparseSpMatDescr_t a_cusparse;
                status = hipsparseCreateCsr(&a_cusparse, nr, nc, nnz,
                                ccsr_v, ccsr_e, ccsr_ev, 
                                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
           
                hipsparseDnMatDescr_t b_cusparse;
                status = hipsparseCreateDnMat(&b_cusparse, nc, sc, sc,
                                  cy_in, HIP_R_32F, HIPSPARSE_ORDER_ROW);

                hipsparseDnMatDescr_t c_cusparse;
                status = hipsparseCreateDnMat(&c_cusparse, nr, sc, sc,
                                      cy_out, HIP_R_32F, HIPSPARSE_ORDER_ROW);
                
                size_t bufferSize = 0;
          
                status = hipsparseSpMM_bufferSize(handle,
                                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                      &alpha,
                                      a_cusparse,
                                      b_cusparse,
                                      &beta,
                                      c_cusparse,
                                      HIP_R_32F,
                                      HIPSPARSE_SPMM_CSR_ALG2,
                                      &bufferSize);

                if (status != HIPSPARSE_STATUS_SUCCESS) return EXIT_FAILURE;
                
                char* externalBuffer = NULL;
                hipMalloc(&externalBuffer, bufferSize);
                
                hipDeviceSynchronize();
                hipEventRecord(event1,0);
            #define ITER (1)
                for(int ik=0;ik<ITER;ik++) {
                    status = hipsparseSpMM(handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha,
                               a_cusparse,
                               b_cusparse,
                               &beta,
                               c_cusparse,
                               HIP_R_32F,
                               HIPSPARSE_SPMM_CSR_ALG2,
                               externalBuffer);
                }

                hipEventRecord(event2,0);
                hipEventSynchronize(event1);
                hipEventSynchronize(event2);
                hipEventElapsedTime(&tot_ms, event1, event2);
                hipDeviceSynchronize();

                if (status != HIPSPARSE_STATUS_SUCCESS) return EXIT_FAILURE;
        	    hipMemcpy(y_out, cy_out, sizeof(FTYPE)*(nr)*sc, hipMemcpyDeviceToHost);

            	hipFree(cy_out); hipFree(cy_in); free(y_out); free(y_in);
                hipFree(externalBuffer);
                // free(loc);
            	fprintf(stdout, "K=%d : nBlocks: %d, nnz: %d, tot_ms: %f ms, GFLOPS: %f \n", sc, nBlock, nnz, tot_ms, (double)ITER*(double)nnz*2*sc/tot_ms/1000000);
            	fprintf(fpo, "%f,", (double)ITER*(double)nnz*2*sc/tot_ms/1000000);
                
                hipFree(ccsr_v), hipFree(ccsr_e); hipFree(ccsr_ev);
                free(csr_v), free(csr_colIdx); free(csr_vals), free(h_csrRowPtr_);
            }
        }
    }
	fclose(fpo);     
}


